/*
 * This file is part of cuAutotools.
 *
 * cuAutotools is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * cuAutotools is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with cuAutotools.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__global__ void kernel(void)
{
}

int main(int argc, char *argv[])
{
  hipError_t d_error = hipSuccess;
  printf("[*] Calling useless kernel...\n");
  kernel<<<1,1>>>();
  d_error = hipGetLastError();
  if(d_error!=hipSuccess)
  {
    fprintf(stderr,"[ERROR] Error running the simples kernel available\n");
    return EXIT_FAILURE;
  }
  printf("[*] Useless kernel called, runned, and exit normally...\n");
  return EXIT_SUCCESS;
}